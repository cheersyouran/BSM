
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h> 
#include <time.h>

#define SIZE_OF_PATTERN 8192
#define SIZE_OF_TARGET 58280548
#define threadNum 8
#define blockNum 256

char Pat[SIZE_OF_PATTERN + 1];
char Tar[SIZE_OF_TARGET + 1];
int Skipping[SIZE_OF_PATTERN + 2][7] = { 0 };
//__constant__ int d_Skipping[(SIZE_OF_PATTERN+2)*7];

void Build_Map(){
	// init Skipping array 
	for (int i = SIZE_OF_PATTERN - 1; i >= 0; i--){
		Skipping[SIZE_OF_PATTERN - i][0] = Pat[i];
		Skipping[SIZE_OF_PATTERN - i - 1][(Pat[i])%5+1] = SIZE_OF_PATTERN - i;
	}
	//bool flag = false;
	int temporary_root;
	int current_processing;
	for (int i = 1; i <= SIZE_OF_PATTERN; i++){
		temporary_root = 0;
		//flag = false;
		for(current_processing = i;current_processing <= SIZE_OF_PATTERN;current_processing ++){
			if (Skipping[temporary_root][((char)Skipping[current_processing][0])%5+1] != 0){
				temporary_root = Skipping[temporary_root][((char)Skipping[current_processing][0])%5+1];
				if (current_processing == SIZE_OF_PATTERN){
					Skipping[temporary_root][6] = 1;
					break;
				}
			}
			else{
				Skipping[temporary_root][((char)Skipping[current_processing][0])%5+1] = current_processing;
				break;
			}
		} 
	}
	Skipping[SIZE_OF_PATTERN][6] = 1;
}

__global__ void MyMethod(char *Tar,int* d_Skipping, int *Output){

	int thd = blockIdx.x*blockDim.x+threadIdx.x;
	int thx = threadIdx.x;
	//int index_in_tar=thx*SIZE_OF_PATTERN;
	int index_in_tar=thd*SIZE_OF_PATTERN;
	int Skip[20];
	for (int i = 0; i < 20; i++){
		Skip[i] = -1;
	}
	//Start searching
	int index_in_pat = 0;
	int index_in_skip = 0;
	int possible_start = 0;
	int tail_Node = 0;
	int test_index = 0;
	int temp_Node = 0;

	if (thd <= SIZE_OF_TARGET/SIZE_OF_PATTERN-1){
		for (index_in_pat = SIZE_OF_PATTERN - 1;index_in_pat >= 0;index_in_pat--){
			//detect skipping number
			if (d_Skipping[index_in_skip*7+(Tar[index_in_tar+index_in_pat])%5+1] != 0){
				index_in_skip = d_Skipping[index_in_skip*7+(Tar[index_in_tar+index_in_pat])%5+1];
				possible_start++;
				if (d_Skipping[index_in_skip*7+6] == 1){
					Skip[tail_Node] = possible_start;
					tail_Node ++;
				}
			}
			else break;			
		}
		while(Skip[temp_Node] != -1){
			if(Tar[index_in_tar+SIZE_OF_PATTERN  + test_index]!=d_Skipping[(SIZE_OF_PATTERN - Skip[temp_Node] - test_index)*7+0]){
				Skip[temp_Node++]=-1;
				test_index = 0;
			}
			else{
				if (Skip[temp_Node] + test_index == SIZE_OF_PATTERN-1){
					//printf("Found it! Start from : %d\n",index_in_tar + 5 - possible_matching[0] );
					Output[thd]=(thd+1)*SIZE_OF_PATTERN - Skip[temp_Node];
					Skip[temp_Node++]=-1;
					test_index = 0;
				}
				else 
					test_index++;
			}
		}	
	}
}

int main (){
	FILE * fp1 = fopen("60.txt", "r");
	FILE * fp2 = fopen("8192.txt", "r");
	fscanf(fp1, "%s", Tar);
	fscanf(fp2, "%s", Pat);
	fclose(fp1);
	fclose(fp2);

	Build_Map();

	int Tar_Size =(SIZE_OF_TARGET + 1)*sizeof(char);
	int Skip_Size = ((SIZE_OF_PATTERN + 2)*7)*sizeof(int);
	int Output[threadNum*blockNum] = {0};
	int Output_Size = threadNum*blockNum*sizeof(int);
	char *d_Tar;
	int *d_Skip;
	int *d_output;

	hipMalloc((void**)&d_Tar,Tar_Size);
	hipMemcpy(d_Tar,Tar,Tar_Size,hipMemcpyHostToDevice);
	hipMalloc((void**)&d_Skip,Skip_Size);
	hipMemcpy(d_Skip,Skipping,Skip_Size,hipMemcpyHostToDevice);

	//cudaMemcpyToSymbol(d_Skipping,Skipping,Skip_Size,0U,cudaMemcpyHostToDevice);

	hipMalloc((void**)&d_output,Output_Size);
	dim3 dimgrid(blockNum,1,1);
	dim3 dimblock(threadNum,1,1);

	float time=0;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	MyMethod<<<dimgrid,dimblock>>>(d_Tar,d_Skip,d_output);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	printf("time is  %f(ms)\n",time/1000);
	
	hipMemcpy(Output,d_output,Output_Size,hipMemcpyDeviceToHost);
	hipFree(d_Tar);
	//cudaFree(d_Skip);
	hipFree(d_output);
	
	for(int i = 0;i<threadNum*blockNum;i++)
		if(Output[i]>0)
			printf("Find it by GPU.%d\n",Output[i]);
	return 0;
}
